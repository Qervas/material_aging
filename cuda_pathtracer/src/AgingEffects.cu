#include "hip/hip_runtime.h"
#include "AgingEffects.cuh"

__device__ float hash(float n) {
    float x = sinf(n) * 43758.5453f;
    return x - floorf(x);
}

__device__ float noise(const Point3f_t& point) {
    Point3f_t p(floorf(point.x), floorf(point.y), floorf(point.z));

    float x = point.x - p.x;
    float y = point.y - p.y;
    float z = point.z - p.z;

    p.x = hash(p.x);
    p.y = hash(p.y);
    p.z = hash(p.z);

    x = x * x * (3.0f - 2.0f * x);
    y = y * y * (3.0f - 2.0f * y);
    z = z * z * (3.0f - 2.0f * z);

    return hash(p.x + p.y * 57.0f + p.z * 113.0f);
}

__device__ Color_t AgingEffects::applyRustEffect(
    const Color_t& base_color,
    const Point3f_t& point,
    const RustParameters& params,
    float noise_value)
{
    // Generate rust pattern
    float pattern = noise(point * params.pattern_scale);
    float rust_amount = pattern * params.oxidation_level;

    // Modify surface properties based on rust
    Color_t rusted_color = lerp(base_color, params.rust_color, rust_amount);

    // Add roughness variation
    float roughness_factor = 1.0f + params.surface_roughness * rust_amount;
    rusted_color *= (1.0f / roughness_factor);

    return rusted_color;
}

__device__ Color_t AgingEffects::applyPaintAging(
    const Color_t& base_color,
    const Point3f_t& point,
    const PaintAgingParameters& params,
    float noise_value)
{
    // Generate crack pattern
    float crack_pattern = noise(point * (1.0f + params.crack_density * 10.0f));

    // Calculate peeling effect
    float peel_threshold = 1.0f - params.peeling_amount;
    float peel_factor = (noise_value > peel_threshold) ? 1.0f : 0.0f;

    // Apply weathering
    Color_t aged_color = base_color * (1.0f - params.weathering * 0.3f);

    // Combine effects
    Color_t final_color = lerp(aged_color, params.underlay_color, peel_factor);

    // Add cracks
    if (crack_pattern < params.crack_density * 0.2f) {
        final_color *= 0.7f;
    }

    return final_color;
}
