#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Window.cuh"
#include "Scene.cuh"
#include "Camera.cuh"
#include "Random.cuh"
#include "raytracer_kernel.cuh"
#include "Material.cuh"
#include "MaterialFactory.cuh"
#include <iostream>
#include <chrono>
#include <algorithm>
#include "Error.cuh"
#include "AgingUI.cuh"

__global__ void updateMetalAgingKernel(Metal_t* material, const RustParameters* params) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        material->setRustParameters(*params);
    }
}

__global__ void updateGlossyAgingKernel(Glossy_t* material, const PaintAgingParameters* params) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        material->setPaintParameters(*params);
    }
}

__global__ void checkAndUpdateMetalKernel(Material_t* material, const RustParameters* params, bool* success) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        if (material->getType() == MaterialType::METAL) {
            Metal_t* metal = static_cast<Metal_t*>(material);
            metal->setRustParameters(*params);
            *success = true;
        } else {
            *success = false;
        }
    }
}

__global__ void checkAndUpdateGlossyKernel(Material_t* material, const PaintAgingParameters* params, bool* success) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        if (material->getType() == MaterialType::GLOSSY) {
            Glossy_t* glossy = static_cast<Glossy_t*>(material);
            glossy->setPaintParameters(*params);
            *success = true;
        } else {
            *success = false;
        }
    }
}


void checkCudaCapabilities() {
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        throw "No CUDA capable devices found";
    }

    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));

    std::cout << "Using GPU: " << prop.name << std::endl;
    std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
}

void initializeCuda() {
    // Initialize CUDA runtime
    CUDA_CHECK(hipSetDevice(0));

    // Create context
    hipFree(0);

    // Check last error
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw hipGetErrorString(error);
    }
}

void updateMetalAging(Sphere_t* sphere, const AgingUI* aging_ui) {
    if (!sphere || !aging_ui) return;

    RustParameters host_params;
    aging_ui->getRustParams(host_params);

    // Create device pointers
    RustParameters* d_params;
    bool* d_success;
    CUDA_CHECK(hipMalloc(&d_params, sizeof(RustParameters)));
    CUDA_CHECK(hipMalloc(&d_success, sizeof(bool)));
    CUDA_CHECK(hipMemcpy(d_params, &host_params, sizeof(RustParameters), hipMemcpyHostToDevice));

    // Update the material parameters using the kernel
    Material_t* material = sphere->getMaterial();
    checkAndUpdateMetalKernel<<<1,1>>>(material, d_params, d_success);
    CUDA_CHECK(hipDeviceSynchronize());

    // Cleanup
    CUDA_CHECK(hipFree(d_params));
    CUDA_CHECK(hipFree(d_success));
}

void updatePaintAging(Sphere_t* sphere, const AgingUI* aging_ui) {
    if (!sphere || !aging_ui) return;

    PaintAgingParameters host_params;
    aging_ui->getPaintParams(host_params);

    // Create device pointers
    PaintAgingParameters* d_params;
    bool* d_success;
    CUDA_CHECK(hipMalloc(&d_params, sizeof(PaintAgingParameters)));
    CUDA_CHECK(hipMalloc(&d_success, sizeof(bool)));
    CUDA_CHECK(hipMemcpy(d_params, &host_params, sizeof(PaintAgingParameters), hipMemcpyHostToDevice));

    // Update the material parameters using the kernel
    Material_t* material = sphere->getMaterial();
    checkAndUpdateGlossyKernel<<<1,1>>>(material, d_params, d_success);
    CUDA_CHECK(hipDeviceSynchronize());

    // Cleanup
    CUDA_CHECK(hipFree(d_params));
    CUDA_CHECK(hipFree(d_success));
}


int main() {
    try {
        // Initialize CUDA with proper error checking
        std::cout << "Initializing CUDA..." << std::endl;
        checkCudaCapabilities();
        initializeCuda();

        // Create window
        std::cout << "Creating window..." << std::endl;
        Window_t window(1920, 1080, "CUDA Ray Tracer");

        // Allocate CUDA memory with error checking
        std::cout << "Allocating GPU memory..." << std::endl;
        float4* d_output = nullptr;
        size_t required_memory = window.getWidth() * window.getHeight() * sizeof(float4);
        CUDA_CHECK(hipMalloc(&d_output, required_memory));

        if (d_output == nullptr) {
            throw "Failed to allocate CUDA memory";
        }

        // Cornell Box dimensions
        const float room_size = 4.0f;
        const float half_size = room_size / 2.0f;

        // Create camera settings struct
        Camera_t::Settings camera_settings{
            90.0f,  // vertical FOV
            window.getWidth() / static_cast<float>(window.getHeight()),  // aspect ratio
            0.0f,   // aperture
            10.0f,  // focus distance
            window.getWidth(),
            window.getHeight(),
            true   // enable_dof
        };

        // Initialize camera
        Camera_t camera(
            Point3f_t(0, 0, half_size * 0.8f),  // position
            Point3f_t(0, 0, -1),                // look at
            Vec3f_t(0, -1, 0),                   // up vector
            camera_settings
        );

        // Create materials on device
        Material_t *d_white_diffuse = nullptr, *d_red_diffuse = nullptr,
                  *d_green_diffuse = nullptr, *d_light = nullptr,
                  *d_glass = nullptr, *d_metal = nullptr, *d_glossy = nullptr,
                    *d_mirror = nullptr;


        MaterialFactory::createMaterialsOnDevice(
            &d_white_diffuse, &d_red_diffuse, &d_green_diffuse,
            &d_light, &d_glass, &d_metal, &d_glossy, &d_mirror
        );

        // Create scene objects
        SceneManager scene_manager;

        // Room walls with subtle colors
        scene_manager.addObject(new Plane_t(Point3f_t(0.0f, -2.0f, 0.0f),
            Vec3f_t(0.0f, 1.0f, 0.0f), d_white_diffuse));  // Floor
        scene_manager.addObject(new Plane_t(Point3f_t(0.0f, 2.0f, 0.0f),
            Vec3f_t(0.0f, -1.0f, 0.0f), d_white_diffuse)); // Ceiling

        scene_manager.addObject(new Plane_t(Point3f_t(-2.0f, 0.0f, 0.0f),
            Vec3f_t(1.0f, 0.0f, 0.0f), d_red_diffuse));    // Left wall
        scene_manager.addObject(new Plane_t(Point3f_t(2.0f, 0.0f, 0.0f),
            Vec3f_t(-1.0f, 0.0f, 0.0f), d_green_diffuse)); // Right wall
        scene_manager.addObject(new Plane_t(
            Point3f_t(0.0f, 0.0f, -2.0f),  // Back wall
            Vec3f_t(0.0f, 0.0f, 1.0f),
            d_mirror
        ));

        scene_manager.addObject(new Plane_t(
            Point3f_t(0.0f, 0.0f, 2.0f),   // Front wall
            Vec3f_t(0.0f, 0.0f, -1.0f),    // Note the opposite normal
            d_mirror
        ));
        // Three main spheres showcasing different materials
        // Center: Large glass sphere (pure specular)
        auto glass_sphere = new Sphere_t(Point3f_t(0.0f, -1.0f, 0.0f),
            0.7f, d_glass);
        scene_manager.addObject(glass_sphere);

        // Left: Glossy sphere (mixed specular/diffuse)
        auto glossy_sphere = new Sphere_t(Point3f_t(-1.2f, -1.2f, -0.5f),
            0.6f, d_glossy);
        scene_manager.addObject(glossy_sphere);

        // Right: Metal sphere (pure specular with color)
        auto metal_sphere = new Sphere_t(Point3f_t(1.2f, -1.2f, -0.5f),
            0.6f, d_metal);
        scene_manager.addObject(metal_sphere);

        // Add some smaller spheres for interaction
        float small_radius = 0.25f;
        auto small_glass = new Sphere_t(Point3f_t(0.0f, -1.4f, -1.2f),
            small_radius, d_glass);
        scene_manager.addObject(small_glass);

        auto small_glossy1 = new Sphere_t(Point3f_t(-0.6f, -1.4f, -1.0f),
            small_radius, d_glossy);
        scene_manager.addObject(small_glossy1);

        auto small_glossy2 = new Sphere_t(Point3f_t(0.6f, -1.4f, -1.0f),
            small_radius, d_glossy);
        scene_manager.addObject(small_glossy2);

        // Lighting setup for better material visualization
        // Main ceiling light
        auto main_light = new Sphere_t(Point3f_t(0.0f, 1.8f, 0.0f),
            0.3f, d_light);
        main_light->makeEmissive(Color_t(1.0f), 15.0f);  // Pure white light
        scene_manager.addObject(main_light);

        // Colored accent lights
        auto blue_light = new Sphere_t(Point3f_t(1.8f, 0.0f, -1.8f),
            0.15f, d_light);
        blue_light->makeEmissive(Color_t(0.2f, 0.2f, 1.0f), 10.0f);
        scene_manager.addObject(blue_light);

        auto orange_light = new Sphere_t(Point3f_t(-1.8f, 0.0f, -1.8f),
            0.15f, d_light);
        orange_light->makeEmissive(Color_t(1.0f, 0.5f, 0.0f), 10.0f);
        scene_manager.addObject(orange_light);

        float tetra_radius = 0.6f;  // Size of the tetrahedron
        Point3f_t tetra_vertices[4] = {
            Point3f_t(0, tetra_radius, 0),                    // Top
            Point3f_t(-tetra_radius, -tetra_radius, tetra_radius),   // Front left
            Point3f_t(tetra_radius, -tetra_radius, tetra_radius),    // Front right
            Point3f_t(0, -tetra_radius, -tetra_radius)        // Back
        };

        // Create and position the tetrahedron
        Vec3f_t tetra_position(-0.6f, 0.0f, -1.0f);
        for (int i = 0; i < 4; ++i) {
            tetra_vertices[i] += tetra_position;
        }

        //tetrahedron faces
        const uint8_t face_indices[4][3] = {
            {0, 2, 1},  // Bottom face
            {0, 1, 3},  // Front face
            {1, 2, 3},  // Right face
            {2, 0, 3}   // Left face
        };

        for (int i = 0; i < 4; ++i) {
            auto triangle = new Triangle_t(
                tetra_vertices[face_indices[i][0]],
                tetra_vertices[face_indices[i][1]],
                tetra_vertices[face_indices[i][2]],
                d_glass
            );
            scene_manager.addObject(triangle);
        }
        // Upload scene to GPU
        scene_manager.uploadToGPU();

        // Initialize GPU data
        initializeGPUData(camera, scene_manager.getDeviceScene());

        // Setup CUDA grid and blocks
        dim3 block(16, 16);
        dim3 grid(
            (window.getWidth() + block.x - 1) / block.x,
            (window.getHeight() + block.y - 1) / block.y
        );

        std::vector<GPUSphere> h_spheres;
        std::vector<GPUPlane> h_planes;
        std::vector<GPUTriangle> h_triangles;


        // Process all objects once
        for (const auto& obj : scene_manager.getHostScene().h_implicit_objects) {
            if (obj->isSphere()) {
                const Sphere_t* sphere = static_cast<const Sphere_t*>(obj);
                GPUSphere gpu_sphere;
                gpu_sphere.center = make_float3(sphere->getCenter().x, sphere->getCenter().y, sphere->getCenter().z);
                gpu_sphere.radius = sphere->getRadius();
                gpu_sphere.material = sphere->getMaterial();
                gpu_sphere.is_emissive = sphere->isEmissive();
                if (sphere->isEmissive()) {
                    Color_t emission = sphere->getEmissionColor() * sphere->getEmissionStrength();
                    gpu_sphere.emission = make_float3(emission.r, emission.g, emission.b);
                } else {
                    gpu_sphere.emission = make_float3(0.0f, 0.0f, 0.0f);
                }
                h_spheres.push_back(gpu_sphere);
            }
            else if (obj->isPlane()) {
                const Plane_t* plane = static_cast<const Plane_t*>(obj);
                GPUPlane gpu_plane;
                gpu_plane.point = make_float3(plane->getPoint().x, plane->getPoint().y, plane->getPoint().z);
                gpu_plane.normal = make_float3(plane->getNormal().x, plane->getNormal().y, plane->getNormal().z);
                gpu_plane.material = plane->getMaterial();
                h_planes.push_back(gpu_plane);
            }
        }

        for (const auto& obj : scene_manager.getHostScene().h_implicit_objects) {
            if (auto triangle = dynamic_cast<const Triangle_t*>(obj)) {
                GPUTriangle gpu_triangle;
                const auto vertices = triangle->getVertices();
                gpu_triangle.v0 = make_float3(vertices[0].x, vertices[0].y, vertices[0].z);
                gpu_triangle.v1 = make_float3(vertices[1].x, vertices[1].y, vertices[1].z);
                gpu_triangle.v2 = make_float3(vertices[2].x, vertices[2].y, vertices[2].z);
                Vec3f_t normal = triangle->getNormalAt(vertices[0]);
                gpu_triangle.normal = make_float3(normal.x, normal.y, normal.z);
                gpu_triangle.material = triangle->getMaterial();
                h_triangles.push_back(gpu_triangle);
            }
        }

        // Upload constants once - only do this once
        int num_spheres = static_cast<int>(h_spheres.size());
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_num_spheres), &num_spheres, sizeof(int)));
        if (num_spheres > 0) {
            CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_spheres), h_spheres.data(), h_spheres.size() * sizeof(GPUSphere)));
        }

        int num_planes = static_cast<int>(h_planes.size());
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_num_planes), &num_planes, sizeof(int)));
        if (num_planes > 0) {
            CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_planes), h_planes.data(), h_planes.size() * sizeof(GPUPlane)));
        }

        int num_triangles = static_cast<int>(h_triangles.size());
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_num_triangles), &num_triangles, sizeof(int)));
        if (num_triangles > 0) {
            CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_triangles), h_triangles.data(),
                                         h_triangles.size() * sizeof(GPUTriangle)));
        }

        initializeGPUData(camera, scene_manager.getDeviceScene());
        // Main rendering loop
        uint32_t frame_count = 0;

        while (window.processEvents()) {
            auto frame_start = std::chrono::high_resolution_clock::now();
            AgingUI* aging_ui = window.getAgingUI();
            aging_ui->render();
            updateMetalAging(metal_sphere, aging_ui);
            updatePaintAging(glossy_sphere, aging_ui);
            // Launch render kernel
            launchRenderKernel(d_output, window.getWidth(), window.getHeight(),
                             frame_count, grid, block);
            CUDA_CHECK(hipGetLastError());

            // Update window with rendered frame
            window.updateScreen(d_output);

            // Calculate and display FPS
            auto frame_end = std::chrono::high_resolution_clock::now();
            auto frame_time = std::chrono::duration_cast<std::chrono::milliseconds>(
                frame_end - frame_start).count();

            std::string title = "CUDA Path Tracer - FPS: " + std::to_string(1000.0f / frame_time);
            window.setTitle(title.c_str());

            frame_count++;
        }

        // Cleanup
        CUDA_CHECK(hipFree(d_output));

        // Cleanup materials before exit
        MaterialFactory::cleanup(
            d_white_diffuse, d_red_diffuse, d_green_diffuse,
            d_light, d_glass, d_metal, d_glossy, d_mirror
        );

    } catch (const std::exception& e) {
        std::cerr << e.what() << std::endl;
        return 1;
    }
    return 0;
}
